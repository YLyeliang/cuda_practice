#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

using namespace std;

static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main() {
    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));

    cout << "Found " << count << " device(s)" << endl;

    for (int i = 0; i < count; ++i) {
        hipDeviceProp_t prop;

        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        cout << "Device name: " << prop.name << endl;
        cout << "Total Memory: " << prop.totalGlobalMem / 1024.0 / 1024.0 << "MB" << endl;
        cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << endl;
        cout << "Compute capability: " << prop.major << "." << prop.minor << endl;

    }

    int result;
    int *devResult;

    HANDLE_ERROR(hipMalloc((void **) &devResult, sizeof(int)));

    add<<<1, 1>>>(7, 8, devResult);

    HANDLE_ERROR(hipMemcpy(&result, devResult, sizeof(int), hipMemcpyDeviceToHost));

    cout << "7 + 8 = " << result << endl;

    hipFree(devResult);

    return 0;
}