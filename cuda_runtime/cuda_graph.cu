//
// Created by fiberhome on 2021/4/1.
//

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "iostream"

using namespace std;


int main() {
    // create the graph
    hipGraph_t graph;
    hipGraphCreate(&graph, 0);

    // create the nodes in two stages, nodes -> dependencies. It also can be specified at node creation
    hipGraphNode_t a;
    hipGraphNode_t b;
    hipGraphAddKernelNode(&a, graph, NULL, 0, &nodeParams);
    hipGraphAddKernelNode(&b, graph, NULL, 0, &nodeParams);

    // set up dependencies
    hipGraphAddDependencies(graph, &a, &b, 1); // A -> B



}
