//
// Created by fiberhome on 2021/4/7.
//
#include "hip/hip_runtime.h"
#include "iostream"

//  __device__  __constant__  __shared__
extern __shared__ float shared[];

// dynamically
extern __shared__ float array[];

__device__ void func()      // __device__ or __global__ function
{
    short *array0 = (short *) array;
    float *array1 = (float *) &array0[128];
    int *array2 = (int *) &array1[64];
}

// Alignment requirement
__device__ void func2()      // __device__ or __global__ function
{
    short *array0 = (short *) array;
    float *array1 = (float *) &array0[127]; // not aligned to 4 bytes, this doesn't work
}


int main() {
    short array0[128];
    float array1[64];
    int array2[256];


}
